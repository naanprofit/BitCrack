#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "windowKernel.h"

#include "secp256k1.cuh"

__device__ static inline bool isZero256(const uint32_t a[8]) {
    for(int i = 0; i < 8; ++i) {
        if(a[i] != 0U) return false;
    }
    return true;
}

__device__ static void setPointInfinity(uint32_t x[8], uint32_t y[8]) {
    for(int i = 0; i < 8; ++i) {
        x[i] = 0xffffffffU;
        y[i] = 0xffffffffU;
    }
}

__device__ static void pointDouble(const uint32_t x[8], const uint32_t y[8],
                                   uint32_t rx[8], uint32_t ry[8]) {
    if(isInfinity(x)) {
        setPointInfinity(rx, ry);
        return;
    }

    uint32_t x2[8];
    uint32_t three_x2[8];
    uint32_t two_y[8];
    uint32_t inv[8];
    uint32_t lambda[8];
    uint32_t lambda2[8];
    uint32_t k[8];

    mulModP(x, x, x2);
    addModP(x2, x2, three_x2);
    addModP(three_x2, x2, three_x2);

    addModP(y, y, two_y);
    invModP(two_y, inv);
    mulModP(three_x2, inv, lambda);

    mulModP(lambda, lambda, lambda2);
    subModP(lambda2, x, rx);
    subModP(rx, x, rx);

    subModP(x, rx, k);
    mulModP(lambda, k, ry);
    subModP(ry, y, ry);
}

__device__ static void pointAdd(const uint32_t ax[8], const uint32_t ay[8],
                                const uint32_t bx[8], const uint32_t by[8],
                                uint32_t rx[8], uint32_t ry[8]) {
    if(isInfinity(ax)) {
        copyBigInt(bx, rx);
        copyBigInt(by, ry);
        return;
    }
    if(isInfinity(bx)) {
        copyBigInt(ax, rx);
        copyBigInt(ay, ry);
        return;
    }
    if(equal(ax, bx) && equal(ay, by)) {
        pointDouble(ax, ay, rx, ry);
        return;
    }

    uint32_t rise[8];
    uint32_t run[8];
    uint32_t inv[8];
    uint32_t lambda[8];
    uint32_t lambda2[8];
    uint32_t k[8];

    subModP(by, ay, rise);
    subModP(bx, ax, run);
    invModP(run, inv);
    mulModP(rise, inv, lambda);

    mulModP(lambda, lambda, lambda2);
    subModP(lambda2, ax, rx);
    subModP(rx, bx, rx);

    subModP(ax, rx, k);
    mulModP(lambda, k, ry);
    subModP(ry, ay, ry);
}

__device__ static void scalarMultiplySmall(const uint32_t bx[8], const uint32_t by[8],
                                           const uint32_t k[8], uint32_t rx[8], uint32_t ry[8]) {
    setPointInfinity(rx, ry);
    uint32_t qx[8];
    uint32_t qy[8];
    copyBigInt(bx, qx);
    copyBigInt(by, qy);
    for(int i = 0; i < 4; ++i) {
        uint32_t word = k[i];
        for(int bit = 0; bit < 32; ++bit) {
            if(word & 1U) {
                uint32_t tx[8];
                uint32_t ty[8];
                pointAdd(rx, ry, qx, qy, tx, ty);
                copyBigInt(tx, rx);
                copyBigInt(ty, ry);
            }
            word >>= 1U;
            uint32_t tx[8];
            uint32_t ty[8];
            pointDouble(qx, qy, tx, ty);
            copyBigInt(tx, qx);
            copyBigInt(ty, qy);
        }
    }
}

__device__ static void scalarMultiplyBase(const uint32_t k[8], uint32_t rx[8], uint32_t ry[8]) {
    GLVScalarSplit split;
    splitScalar(k, split);

    uint32_t r1x[8];
    uint32_t r1y[8];
    scalarMultiplySmall(_GX, _GY, split.k1, r1x, r1y);
    if(split.k1Neg) {
        uint32_t ny[8];
        negModP(r1y, ny);
        copyBigInt(ny, r1y);
    }

    if(isZero256(split.k2)) {
        copyBigInt(r1x, rx);
        copyBigInt(r1y, ry);
        return;
    }

    uint32_t base2x[8];
    uint32_t base2y[8];
    copyBigInt(_GX, base2x);
    copyBigInt(_GY, base2y);
    mulModP(base2x, _BETA, base2x);

    uint32_t r2x[8];
    uint32_t r2y[8];
    scalarMultiplySmall(base2x, base2y, split.k2, r2x, r2y);
    if(split.k2Neg) {
        uint32_t ny[8];
        negModP(r2y, ny);
        copyBigInt(ny, r2y);
    }

    pointAdd(r1x, r1y, r2x, r2y, rx, ry);
}

// GPU kernel performing a grid-stride loop over scalars ``k`` and extracting
// window fragments from the x-coordinate of ``k * G``. Matching fragments are
// appended to ``out_buf`` using an atomic counter.
extern "C" __global__
void windowKernel(uint64_t start_k, uint64_t range_len, uint32_t ws,
                  const uint32_t* offsets, uint32_t offsets_count,
                  uint32_t mask, const uint32_t* target_frags,
                  MatchRecord* out_buf, uint32_t* out_count) {
    (void)ws; // window size is encoded in mask on device
    uint64_t idx    = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;
    for(uint64_t i = idx; i < range_len; i += stride) {
        uint64_t k = start_k + i;
        uint32_t X[8], Y[8];
        scalarMultiplyBase(reinterpret_cast<const uint32_t*>(&k), X, Y);
        for(uint32_t j = 0; j < offsets_count; ++j) {
            uint32_t off  = offsets[j];
            uint32_t word = off >> 5;
            uint32_t bit  = off & 31u;
            uint32_t frag = 0;
            if(word < 8) {
                frag = X[word] >> bit;
                if(bit && word + 1 < 8) {
                    frag |= X[word + 1] << (32 - bit);
                }
                frag &= mask;
                if(frag == target_frags[j]) {
                    uint32_t pos = atomicAdd(out_count, 1u);
                    out_buf[pos] = { off, frag, k };
                }
            }
        }
    }
}

// Host wrapper used to launch ``windowKernel`` with basic error checking.
extern "C" void launchWindowKernel(dim3 grid, dim3 block,
                                   uint64_t start_k, uint64_t range_len,
                                   uint32_t ws, const uint32_t* offsets,
                                   uint32_t offsets_count, uint32_t mask,
                                   const uint32_t* target_frags,
                                   MatchRecord* out_buf,
                                   uint32_t* out_count) {
    windowKernel<<<grid, block>>>(start_k, range_len, ws, offsets,
                                  offsets_count, mask, target_frags,
                                  out_buf, out_count);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        exit(1);
    }
}

