#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include "secp256k1.cuh"
#include "windowKernel.h"

// Simple CUDA error checking macro used throughout this file.
#define CUDA_CHECK(call) do { \
    hipError_t err = (call); \
    if(err != hipSuccess) { \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
    } \
} while(0)

// -- EC helper routines ------------------------------------------------------

__device__ static inline bool isZero256(const uint32_t a[8]) {
    for(int i = 0; i < 8; ++i) {
        if(a[i] != 0U) return false;
    }
    return true;
}

__device__ static void setPointInfinity(uint32_t x[8], uint32_t y[8]) {
    for(int i = 0; i < 8; ++i) {
        x[i] = 0xffffffffU;
        y[i] = 0xffffffffU;
    }
}

__device__ static void pointDouble(const uint32_t x[8], const uint32_t y[8],
                                   uint32_t rx[8], uint32_t ry[8]) {
    if(isInfinity(x)) {
        setPointInfinity(rx, ry);
        return;
    }

    uint32_t x2[8];
    uint32_t three_x2[8];
    uint32_t two_y[8];
    uint32_t inv[8];
    uint32_t lambda[8];
    uint32_t lambda2[8];
    uint32_t k[8];

    mulModP(x, x, x2);
    addModP(x2, x2, three_x2);
    addModP(three_x2, x2, three_x2);

    addModP(y, y, two_y);
    invModP(two_y, inv);
    mulModP(three_x2, inv, lambda);

    mulModP(lambda, lambda, lambda2);
    subModP(lambda2, x, rx);
    subModP(rx, x, rx);

    subModP(x, rx, k);
    mulModP(lambda, k, ry);
    subModP(ry, y, ry);
}

__device__ static void pointAdd(const uint32_t ax[8], const uint32_t ay[8],
                                const uint32_t bx[8], const uint32_t by[8],
                                uint32_t rx[8], uint32_t ry[8]) {
    if(isInfinity(ax)) {
        copyBigInt(bx, rx);
        copyBigInt(by, ry);
        return;
    }
    if(isInfinity(bx)) {
        copyBigInt(ax, rx);
        copyBigInt(ay, ry);
        return;
    }
    if(equal(ax, bx) && equal(ay, by)) {
        pointDouble(ax, ay, rx, ry);
        return;
    }

    uint32_t rise[8];
    uint32_t run[8];
    uint32_t inv[8];
    uint32_t lambda[8];
    uint32_t lambda2[8];
    uint32_t k[8];

    subModP(by, ay, rise);
    subModP(bx, ax, run);
    invModP(run, inv);
    mulModP(rise, inv, lambda);

    mulModP(lambda, lambda, lambda2);
    subModP(lambda2, ax, rx);
    subModP(rx, bx, rx);

    subModP(ax, rx, k);
    mulModP(lambda, k, ry);
    subModP(ry, ay, ry);
}

__device__ static void scalarMultiplySmall(const uint32_t bx[8], const uint32_t by[8],
                                           const uint32_t k[8], uint32_t rx[8], uint32_t ry[8]) {
    setPointInfinity(rx, ry);
    uint32_t qx[8];
    uint32_t qy[8];
    copyBigInt(bx, qx);
    copyBigInt(by, qy);
    for(int i = 0; i < 4; ++i) {
        uint32_t word = k[i];
        for(int bit = 0; bit < 32; ++bit) {
            if(word & 1U) {
                uint32_t tx[8];
                uint32_t ty[8];
                pointAdd(rx, ry, qx, qy, tx, ty);
                copyBigInt(tx, rx);
                copyBigInt(ty, ry);
            }
            word >>= 1U;
            uint32_t tx[8];
            uint32_t ty[8];
            pointDouble(qx, qy, tx, ty);
            copyBigInt(tx, qx);
            copyBigInt(ty, qy);
        }
    }
}

__device__ static void scalarMultiplyBase(const uint32_t k[8], uint32_t rx[8], uint32_t ry[8]) {
    GLVScalarSplit split;
    splitScalar(k, split);

    uint32_t r1x[8];
    uint32_t r1y[8];
    scalarMultiplySmall(_GX, _GY, split.k1, r1x, r1y);
    if(split.k1Neg) {
        uint32_t ny[8];
        negModP(r1y, ny);
        copyBigInt(ny, r1y);
    }

    if(isZero256(split.k2)) {
        copyBigInt(r1x, rx);
        copyBigInt(r1y, ry);
        return;
    }

    uint32_t base2x[8];
    uint32_t base2y[8];
    copyBigInt(_GX, base2x);
    copyBigInt(_GY, base2y);
    mulModP(base2x, _BETA, base2x);

    uint32_t r2x[8];
    uint32_t r2y[8];
    scalarMultiplySmall(base2x, base2y, split.k2, r2x, r2y);
    if(split.k2Neg) {
        uint32_t ny[8];
        negModP(r2y, ny);
        copyBigInt(ny, r2y);
    }

    pointAdd(r1x, r1y, r2x, r2y, rx, ry);
}

__device__ static inline void point_mul_G(const uint32_t k[8], uint32_t X[8], uint32_t Y[8]) {
    scalarMultiplyBase(k, X, Y);
}

// -----------------------------------------------------------------------------

extern "C" __global__ void windowKernel(uint64_t start_k,
                                         uint64_t range_len,
                                         uint32_t ws,
                                         const uint32_t *offsets,
                                         uint32_t offsets_count,
                                         uint32_t mask,
                                         const uint32_t *target_frags,
                                         MatchRecord *out_buf,
                                         uint32_t *out_count) {
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = gridDim.x * blockDim.x;

    // Grid-stride loop over the scalar range.
    for(uint64_t idx = tid; idx < range_len; idx += stride) {
        uint64_t k = start_k + idx;

        // Convert ``k`` into the little-endian 256-bit representation expected
        // by the secp256k1 routines.
        uint32_t scalar[8] = {0};
        scalar[0] = (uint32_t)(k & 0xffffffffULL);
        scalar[1] = (uint32_t)(k >> 32);

        uint32_t X[8];
        uint32_t Y[8];
        point_mul_G(scalar, X, Y);

        // Examine each requested window.
        for(uint32_t i = 0; i < offsets_count; ++i) {
            uint32_t off = offsets[i];
            uint32_t word = off >> 5;
            uint32_t bit  = off & 31U;
            uint64_t val = ((uint64_t)X[word]) >> bit;
            if(bit && word < 7) {
                val |= ((uint64_t)X[word + 1]) << (32 - bit);
            }
            uint32_t frag = (uint32_t)val & mask;
            if(frag == target_frags[i]) {
                uint32_t outIdx = atomicAdd(out_count, 1u);
                out_buf[outIdx].offset   = off;
                out_buf[outIdx].fragment = frag;
                out_buf[outIdx].k        = k;
            }
        }
    }
}

extern "C" void launchWindowKernel(dim3 grid,
                                   dim3 block,
                                   uint64_t start_k,
                                   uint64_t range_len,
                                   uint32_t ws,
                                   const uint32_t *offsets,
                                   uint32_t offsets_count,
                                   uint32_t mask,
                                   const uint32_t *target_frags,
                                   MatchRecord *out_buf,
                                   uint32_t *out_count) {
    // Launch the kernel and check for launch/runtime errors.
    windowKernel<<<grid, block>>>(start_k, range_len, ws, offsets,
                                  offsets_count, mask, target_frags,
                                  out_buf, out_count);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

